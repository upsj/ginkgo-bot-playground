/*******************************<GINKGO LICENSE>******************************
Copyright (c) 2017-2021, the Ginkgo authors
All rights reserved.

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions
are met:

1. Redistributions of source code must retain the above copyright
notice, this list of conditions and the following disclaimer.

2. Redistributions in binary form must reproduce the above copyright
notice, this list of conditions and the following disclaimer in the
documentation and/or other materials provided with the distribution.

3. Neither the name of the copyright holder nor the names of its
contributors may be used to endorse or promote products derived from
this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS
IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED
TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT
HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,
SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT
LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
******************************<GINKGO LICENSE>*******************************/

#include "core/multigrid/amgx_pgm_kernels.hpp"


#include <memory>


#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <thrust/tuple.h>


#include <ginkgo/core/base/exception_helpers.hpp>
#include <ginkgo/core/base/math.hpp>
#include <ginkgo/core/multigrid/amgx_pgm.hpp>


#include "core/components/fill_array.hpp"
#include "core/components/prefix_sum.hpp"
#include "core/matrix/csr_builder.hpp"
#include "core/matrix/csr_kernels.hpp"
#include "cuda/base/cusparse_bindings.hpp"
#include "cuda/base/math.hpp"
#include "cuda/base/types.hpp"
#include "cuda/components/atomic.cuh"
#include "cuda/components/reduction.cuh"
#include "cuda/components/thread_ids.cuh"


namespace gko {
namespace kernels {
namespace cudaa {
/**
 * @brief The AMGX_PGM solver namespace.
 *
 * @ingroup amgx_pgm
 */
namespace amgx_pgm {


constexpr int default_block_size = 512;


#include "common/multigrid/amgx_pgm_kernels.hpp.inc"


template <typename IndexType>
void match_edge(std::shared_ptr<const CudaExecutor> exec,
                const Array<IndexType> &strongest_neighbor,
                Array<IndexType> &agg)
{
    const auto num = agg.get_num_elems();
    const dim3 grid(ceildiv(num, default_block_size));
    kernel::match_edge_kernel<<<grid, default_block_size>>>(
        num, strongest_neighbor.get_const_data(), agg.get_data());
}

GKO_INSTANTIATE_FOR_EACH_INDEX_TYPE(GKO_DECLARE_AMGX_PGM_MATCH_EDGE_KERNEL);


template <typename IndexType>
void count_unagg(std::shared_ptr<const CudaExecutor> exec,
                 const Array<IndexType> &agg, IndexType *num_unagg)
{
    Array<IndexType> active_agg(exec, agg.get_num_elems());
    const dim3 grid(ceildiv(active_agg.get_num_elems(), default_block_size));
    kernel::activate_kernel<<<grid, default_block_size>>>(
        active_agg.get_num_elems(), agg.get_const_data(),
        active_agg.get_data());
    *num_unagg = reduce_add_array(exec, active_agg.get_num_elems(),
                                  active_agg.get_const_data());
}

GKO_INSTANTIATE_FOR_EACH_INDEX_TYPE(GKO_DECLARE_AMGX_PGM_COUNT_UNAGG_KERNEL);


template <typename IndexType>
void renumber(std::shared_ptr<const CudaExecutor> exec, Array<IndexType> &agg,
              IndexType *num_agg)
{
    const auto num = agg.get_num_elems();
    Array<IndexType> agg_map(exec, num + 1);
    const dim3 grid(ceildiv(num, default_block_size));
    kernel::fill_agg_kernel<<<grid, default_block_size>>>(
        num, agg.get_const_data(), agg_map.get_data());
    components::prefix_sum(exec, agg_map.get_data(), agg_map.get_num_elems());
    kernel::renumber_kernel<<<grid, default_block_size>>>(
        num, agg_map.get_const_data(), agg.get_data());
    *num_agg = exec->copy_val_to_host(agg_map.get_const_data() + num);
}

GKO_INSTANTIATE_FOR_EACH_INDEX_TYPE(GKO_DECLARE_AMGX_PGM_RENUMBER_KERNEL);


template <typename ValueType, typename IndexType>
void find_strongest_neighbor(
    std::shared_ptr<const CudaExecutor> exec,
    const matrix::Csr<ValueType, IndexType> *weight_mtx,
    const matrix::Diagonal<ValueType> *diag, Array<IndexType> &agg,
    Array<IndexType> &strongest_neighbor)
{
    const auto num = agg.get_num_elems();
    const dim3 grid(ceildiv(num, default_block_size));
    kernel::find_strongest_neighbor_kernel<<<grid, default_block_size>>>(
        num, weight_mtx->get_const_row_ptrs(), weight_mtx->get_const_col_idxs(),
        weight_mtx->get_const_values(), diag->get_const_values(),
        agg.get_data(), strongest_neighbor.get_data());
}

GKO_INSTANTIATE_FOR_EACH_NON_COMPLEX_VALUE_AND_INDEX_TYPE(
    GKO_DECLARE_AMGX_PGM_FIND_STRONGEST_NEIGHBOR);

template <typename ValueType, typename IndexType>
void assign_to_exist_agg(std::shared_ptr<const CudaExecutor> exec,
                         const matrix::Csr<ValueType, IndexType> *weight_mtx,
                         const matrix::Diagonal<ValueType> *diag,
                         Array<IndexType> &agg,
                         Array<IndexType> &intermediate_agg)
{
    const auto num = agg.get_num_elems();
    const dim3 grid(ceildiv(num, default_block_size));
    if (intermediate_agg.get_num_elems() > 0) {
        // determinstic kernel
        kernel::assign_to_exist_agg_kernel<<<grid, default_block_size>>>(
            num, weight_mtx->get_const_row_ptrs(),
            weight_mtx->get_const_col_idxs(), weight_mtx->get_const_values(),
            diag->get_const_values(), agg.get_const_data(),
            intermediate_agg.get_data());
        // Copy the intermediate_agg to agg
        agg = intermediate_agg;
    } else {
        // undeterminstic kernel
        kernel::assign_to_exist_agg_kernel<<<grid, default_block_size>>>(
            num, weight_mtx->get_const_row_ptrs(),
            weight_mtx->get_const_col_idxs(), weight_mtx->get_const_values(),
            diag->get_const_values(), agg.get_data());
    }
}

GKO_INSTANTIATE_FOR_EACH_NON_COMPLEX_VALUE_AND_INDEX_TYPE(
    GKO_DECLARE_AMGX_PGM_ASSIGN_TO_EXIST_AGG);


}  // namespace amgx_pgm
}  // namespace cudaa
}  // namespace kernels
}  // namespace gko
